#include "hip/hip_runtime.h"
// -*- C++ -*-
// -*- coding: utf-8 -*-
//
// (c) 2013-2019 parasim inc
// (c) 2010-2019 california institute of technology
// all rights reserved
//
// Author(s): Hailiang Zhang, Lijun Zhu


// declarations 
#include "cudaGaussian.h"
// cuda utilities
#include <pyre/hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// cuda kernel declarations
namespace cudaGaussian_kernels {
    // sample 
    template<typename real_type> 
    __global__ void _sample(hiprandState_t * curand_states, 
        real_type * const theta, const size_t samples, const size_t parameters,  
        const size_t idx_begin, const size_t idx_end, 
        const real_type mean, const real_type sigma);

    template <>
    __global__ void _sample<double>(hiprandState_t * curand_states,
        double * const theta, const size_t samples, const size_t parameters,
        const size_t idx_begin, const size_t idx_end,
        const double mean, const double sigma);

    template <>
    __global__ void _sample<float>(hiprandState_t * curand_states,
        float * const theta, const size_t samples, const size_t parameters,
        const size_t idx_begin, const size_t idx_end,
        const float mean, const float sigma);

    // log pdf
    template<typename real_type> 
    __global__ void _logpdf(const real_type * const theta, real_type * const probability,
        const size_t samples, const size_t parameters, 
        const size_t idx_begin, const size_t idx_end, 
        const real_type mean, const real_type sigma);
}  
 
// generate random samples
template<typename real_type> 
void altar::cuda::distributions::cudaGaussian::
sample(real_type * const theta, const size_t samples, const size_t parameters, 
                    const size_t idx_begin, const size_t idx_end,
                    const real_type mean, const real_type sigma,
                    hipStream_t stream)
{
    // determine the block/grid size
    // one thread for one sample
    int blockSize = NTHREADS;
    int gridSize = IDIVUP(samples, blockSize);
    
    // allocate 
    hiprandState_t *curand_states;
    cudaSafeCall(hipMalloc((void**)&curand_states, blockSize*gridSize*sizeof(hiprandState)));
    
    // call cuda kernels
    cudaGaussian_kernels::_sample<real_type><<<gridSize, blockSize, 0, stream>>>(curand_states, 
        theta, samples, parameters, idx_begin, idx_end, mean, sigma);
    cudaCheckError("cudaGaussian::random generation error");

    cudaSafeCall(hipFree(curand_states));
} 

// explicit instantiation
template void altar::cuda::distributions::cudaGaussian::sample<float>(float * const, const size_t, const size_t, 
                    const size_t, const size_t, const float, const float, hipStream_t);
template void altar::cuda::distributions::cudaGaussian::sample<double>(double * const, const size_t, const size_t, 
                    const size_t, const size_t, const double, const double, hipStream_t);

                    
// compute log probability
template <typename real_type>
void altar::cuda::distributions::cudaGaussian::
logpdf(const real_type * const theta, real_type * const probability, 
                    const size_t samples, const size_t parameters, 
                    const size_t idx_begin, const size_t idx_end,
                    const real_type mean, const real_type sigma,
                    hipStream_t stream)
{
    int blockSize = NTHREADS;
    int gridSize = IDIVUP(samples, blockSize);

    // call cuda kernels
    cudaGaussian_kernels::_logpdf<real_type><<<gridSize, blockSize, 0, stream>>>(
        theta, probability, samples, parameters, idx_begin, idx_end, mean, sigma);
    cudaCheckError("cudaGaussian:: log_pdf error");
}

// explicit instantiation
template void altar::cuda::distributions::cudaGaussian::logpdf<float>(const float * const, float * const, const size_t, const size_t, 
                    const size_t, const size_t, const float, const float, hipStream_t);
template void altar::cuda::distributions::cudaGaussian::logpdf<double>(const double * const, double * const, const size_t, const size_t, 
                    const size_t, const size_t, const double, const double, hipStream_t);

// put explicit specialization in a namespace due to a bug in gcc6
namespace cudaGaussian_kernels {

//random_generation_kernel
// double precision version
template <>
__global__ void
_sample<double>(hiprandState_t * curand_states, 
    double * const theta, const size_t samples, const size_t parameters, 
    const size_t idx_begin, const size_t idx_end, 
    const double mean, const double sigma)
{
    int sample = blockIdx.x*blockDim.x + threadIdx.x;
    if (sample >= samples) return;
    
    // initialize seeds for each thread    
    unsigned long long seed = (unsigned long long) clock64(); 
    hiprand_init(seed, sample, 0, &curand_states[sample]); 
    
    // get the theta pointer for each sample 
    double * theta_sample = theta + sample*parameters;
    
    // generate samples from idx_begin to idx_end 
    for (int i=idx_begin; i<idx_end; ++i)
    {
        theta_sample[i] = hiprand_normal_double(&curand_states[sample])*sigma + mean;
    }
}

//single precision version
template <>
__global__ void
_sample<float>(hiprandState_t * curand_states, 
    float * const theta, const size_t samples, const size_t parameters, 
    const size_t idx_begin, const size_t idx_end, 
    const float mean, const float sigma)
{
    int sample = blockIdx.x*blockDim.x + threadIdx.x;
    if (sample >= samples) return;
    
    // initialize seeds for each thread    
    unsigned long long seed = (unsigned long long) clock64(); 
    hiprand_init(seed, sample, 0, &curand_states[sample]); 
    
    // get the theta pointer for each sample 
    float * theta_sample = theta + sample*parameters;
    
    // generate samples from idx_begin to idx_end 
    for (int i=idx_begin; i<idx_end; ++i)
    {
        theta_sample[i] = hiprand_normal(&curand_states[sample])*sigma + mean;
    }
}

} // of namespace cudaGaussian_kernels

//log_pdf kernel
template <typename real_type>
__global__ void
cudaGaussian_kernels::
_logpdf(const real_type * const theta, real_type * const probability, const size_t samples, const size_t parameters, 
        const size_t idx_begin, const size_t idx_end, const real_type mean, const real_type sigma)
{
    // get the thread/sample id
    int sample = blockIdx.x*blockDim.x + threadIdx.x;
    if (sample >= samples) return;

    // get the theta pointer
    const real_type * theta_sample = theta + sample*parameters;
    real_type log_pdf = 0.0;
    real_type c1 = -log( sigma * sqrt(2.*PI) );
    real_type c2 = 0.5/(sigma*sigma);
    
    //  
    for (int i=idx_begin; i<idx_end; ++i)
    {
        real_type mtmp = theta_sample[i]-mean;
        log_pdf += c1-mtmp*mtmp*c2;
    }
        
    probability[sample] += log_pdf;
}

// end of file
